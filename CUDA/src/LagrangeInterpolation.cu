
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void interpolate(float * x, float * y, float a, float * k, int n){
    int i,j;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float ss[100], ts[100], ks[100];
    
    if(i<n)
    {
        ss[i]=1;
        ts[i]=1;
        __syncthreads();

        for(j=0;j<n;j++)
        {
            if(j!=i)
            {
                ss[i]=ss[i]*(a-x[j]);
                ts[i]=ts[i]*(x[i]-x[j]);
            }
        }

        ks[i]=(((ss[i])/(ts[i]))*y[i]);
	__syncthreads();
	if(i==0){
	    for(i=0;i<(n*n);i++){
	        *k += ks[i];
	    }
	}
    }
}

int main()
{
    float *x, *y, *d_x, *d_y;
    float a, k, *d_k;  
    size_t size = 100 * sizeof(float);
    hipEvent_t start, stop;
    
    x = new float[100];
    y = new float[100];
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_k, sizeof(float));
    hipEventCreate(&start);
    hipEventCreate(&stop);


    int N,i,d=1;
    printf("\n\n Enter the number of the terms of the table: ");
    scanf("%d",&N);
    printf("%d", N);
    printf("\n\n Enter the respective values of the variables x and y: \n");
    for(i=0; i<N; i++)
    {
        scanf ("%f",&x[i]);
        scanf("%f",&y[i]);
    }
    printf("\n\n The table you entered is as follows :\n\n");

    for(i=0; i<N; i++)
    {
        printf("%0.3f\t%0.3f",x[i],y[i]);
        printf("\n");
    }
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    //while(d==1)
    //{
        printf(" \n\n\n Enter the value of the x to find the respective value of y\n\n\n");
        scanf("%f",&a);
	printf("%f\n",a);

	int threads_per_block = 32; // A 16 x 16 block threads
  	int number_of_blocks = N/threads_per_block + 1;
	//dim3 threads_per_block(3, 3); // A 16 x 16 block threads
  	//dim3 number_of_blocks(N/threads_per_block.x + 1, N/threads_per_block.y + 1);
	hipEventRecord(start);
	interpolate<<<number_of_blocks,threads_per_block>>>(d_x, d_y, a, d_k, N);

        hipError_t err;
        err = hipGetLastError(); // `cudaGetLastError` will return the error from above.
        if (err != hipSuccess)
        { 
            printf("Error: %s\n", hipGetErrorString(err));
        }
	hipEventSynchronize(stop);
  	float milliseconds = 0;
  	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(&k, d_k, sizeof(float), hipMemcpyDeviceToHost);

        printf("\n\n The respective value of the variable y is: %f\n", k);
	printf(" Elapsed time in milliseconds: %f\n", milliseconds);
        printf("\n\n Do you want to continue?\n\n Press 1 to continue and any other key to exit");
        scanf("%d",&d);
    //}

    delete [] x;
    delete [] y;
    hipFree(d_x);
    hipFree(d_y);
}
